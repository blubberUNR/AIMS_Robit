#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include ""

#include "kernel.h"


__global__ void fd_classify(unsigned char* image, unsigned char* strictMask, unsigned char* looseMask, decisionMap* strictMaps, decisionMap* looseMaps, int width, int height)
{
	//get the yIndex and xIndex associated with this pixel
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//guard against pixels outside of the image
	if (xIndex < width && yIndex < height)
	{
		//get the image index associated with this pixel
		int i = xIndex + yIndex * width;

		//get the current pixel's values
		int r = image[i * 3 + 2] / (int)(256.0f / (float)NUM_BINS);
		int g = image[i * 3 + 1] / (int)(256.0f / (float)NUM_BINS);
		int b = image[i * 3] / (int)(256.0f / (float)NUM_BINS);
		int y = (int)((float)r * 0.3f + (float)g * 0.6f + (float)b * 0.1f);
		int cr = 0;
		int cb = 0;
		if (r + g + b != 0)
		{
			cr = r * NUM_BINS / (r + g + b);
			cb = b * NUM_BINS / (r + g + b);
		}

		//if the pixel is strictly in the foreground...
		if (strictMaps[i].decision[cr * 3] || strictMaps[i].decision[cb * 3 + 1] || strictMaps[i].decision[y * 3 + 2])
		{
			//set the mask color to white
			strictMask[i] = 255;
		}
		else //else if the pixel is in the background...
		{
			//set the mask color to black
			strictMask[i] = 0;
		}

		//if the pixel is loosely in the foreground...
		if (looseMaps[i].decision[cr * 3] || looseMaps[i].decision[cb * 3 + 1] || looseMaps[i].decision[y * 3 + 2])
		{
			//set the mask color to white
			looseMask[i] = 255;
		}
		else //else if the pixel is in the background...
		{
			//set the mask color to black
			looseMask[i] = 0;
		}
	}
}

__global__ void fd_update(unsigned char* image, energy* energies, int width, int height, int yOffset)
{
	//get the yIndex and xIndex associated with this pixel
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = (blockIdx.y + yOffset) * blockDim.y + threadIdx.y;

	//guard against pixels outside of the image
	if (xIndex < width && yIndex < height)
	{
		//get the image index associated with this pixel
		int i = xIndex + yIndex * width;

		//get the current pixel's values
		int r = image[i * 3 + 2] / (int)(256.0f / (float)NUM_BINS);
		int g = image[i * 3 + 1] / (int)(256.0f / (float)NUM_BINS);
		int b = image[i * 3] / (int)(256.0f / (float)NUM_BINS);
		int y = (int)((float)r * 0.3f + (float)g * 0.6f + (float)b * 0.1f);
		int cr = 0;
		int cb = 0;
		if (r + g + b != 0)
		{
			cr = r * NUM_BINS / (r + g + b);
			cb = b * NUM_BINS / (r + g + b);
		}

		//Increment this pixels counters
		energies[i].count[cr * 3]++;
		energies[i].count[cb * 3 + 1]++;
		energies[i].count[y * 3 + 2]++;
		energies[i].total++;
	}
}

__global__ void fd_train(energy* energies, histogram* histograms, decisionMap* strictMaps, decisionMap* looseMaps, int width, int height, int xOffset, int yOffset, float alpha, float strictThreshold, int strictRange, float looseThreshold, int looseRange)
{
	//get the yIndex and xIndex associated with this pixel
	int xIndex = (blockIdx.x + xOffset) * blockDim.x + threadIdx.x;
	int yIndex = (blockIdx.y + yOffset) * blockDim.y + threadIdx.y;

	//guard against pixels outside of the image
	if (xIndex < width && yIndex < height)
	{
		//get the image index associated with this pixel
		int i = xIndex + yIndex * width;

		//Get the sum of each pixel counter
		int cSum = energies[i].total;
		energies[i].total = 0;

		//Get a copy of the relevant histogram in local memory
		histogram h;
		for (int x = 0; x < NUM_BINS; x++)
		{
			h.bin[x * 3] = histograms[i].bin[x * 3];
			h.bin[x * 3 + 1] = histograms[i].bin[x * 3 + 1];
			h.bin[x * 3 + 2] = histograms[i].bin[x * 3 + 2];
		}

		//Set the new bin value based on alpha and the normalized count for this pixel, sum each new bin value, and reset the count for this pixel
		float crSum = 0.0f;
		float cbSum = 0.0f;
		float ySum = 0.0f;
		for (int x = 0; x < NUM_BINS; x++)
		{
			h.bin[x * 3] = ((float)energies[i].count[x * 3] / (float)cSum) * alpha + h.bin[x * 3] * (1.0f - alpha);
			crSum += h.bin[x * 3];
			energies[i].count[x * 3] = 0;

			h.bin[x * 3 + 1] = ((float)energies[i].count[x * 3 + 1] / (float)cSum) * alpha + h.bin[x * 3 + 1] * (1.0f - alpha);
			cbSum += h.bin[x * 3 + 1];
			energies[i].count[x * 3 + 1] = 0;

			h.bin[x * 3 + 2] = ((float)energies[i].count[x * 3 + 2] / (float)cSum) * alpha + h.bin[x * 3 + 2] * (1.0f - alpha);
			ySum += h.bin[x * 3 + 2];
			energies[i].count[x * 3 + 2] = 0;
		}

		//Divide each bin by bSum to normalize them, and then find the global max
		float crMax = 0.0f;
		float crMin = 1.0f;
		float cbMax = 0.0f;
		float cbMin = 1.0f;
		float yMax = 0.0f;
		float yMin = 1.0f;
		for (int x = 0; x < NUM_BINS; x++)
		{
			h.bin[x * 3] /= crSum;//scan[0]
			if (h.bin[x * 3] > crMax)
				crMax = h.bin[x * 3];
			if (h.bin[x * 3] < crMin)
				crMin = h.bin[x * 3];

			h.bin[x * 3 + 1] /= cbSum;//scan[1]
			if (h.bin[x * 3 + 1] > cbMax)
				cbMax = h.bin[x * 3 + 1];
			if (h.bin[x * 3 + 1] < cbMin)
				cbMin = h.bin[x * 3 + 1];

			h.bin[x * 3 + 2] /= ySum;//scan[2]
			if (h.bin[x * 3 + 2] > yMax)
				yMax = h.bin[x * 3 + 2];
			if (h.bin[x * 3 + 2] < yMin)
				yMin = h.bin[x * 3 + 2];
		}

		//Calculate the strict and loose thresholds for each classification criteria
		float crStrictThreshold = crMax * strictThreshold + crMin * (1.0f - strictThreshold);
		float cbStrictThreshold = cbMax * strictThreshold + cbMin * (1.0f - strictThreshold);
		float yStrictThreshold = yMax * strictThreshold + yMin * (1.0f - strictThreshold);
		float crLooseThreshold = crMax * looseThreshold + crMin * (1.0f - looseThreshold);
		float cbLooseThreshold = cbMax * looseThreshold + cbMin * (1.0f - looseThreshold);
		float yLooseThreshold = yMax * looseThreshold + yMin * (1.0f - looseThreshold);

		//Create the new decision maps on local memory
		decisionMap strictMap;
		decisionMap looseMap;

		//Perform the first strict decision map generation pass
		int crCount = 0;
		int cbCount = 0;
		int yCount = 0;
		for (int x = 0; x < NUM_BINS; x++)
		{
			if (h.bin[x * 3] > crStrictThreshold)
				crCount = strictRange;
			else
				crCount--;

			if (crCount > 0)
				strictMap.decision[x * 3] = false;
			else
				strictMap.decision[x * 3] = true;

			if (h.bin[x * 3 + 1] > cbStrictThreshold)
				cbCount = strictRange;
			else
				cbCount--;

			if (cbCount > 0)
				strictMap.decision[x * 3 + 1] = false;
			else
				strictMap.decision[x * 3 + 1] = true;

			if (h.bin[x * 3 + 2] > yStrictThreshold)
				yCount = strictRange;
			else
				yCount--;

			if (yCount > 0)
				strictMap.decision[x * 3 + 2] = false;
			else
				strictMap.decision[x * 3 + 2] = true;
		}

		//Perform the second strict decision map generation pass
		crCount = 0;
		cbCount = 0;
		yCount = 0;
		for (int x = NUM_BINS - 1; x >= 0; x--)
		{
			if (h.bin[x * 3] > crStrictThreshold)
				crCount = strictRange;
			else
				crCount--;

			if (crCount > 0)
				strictMap.decision[x * 3] = false;

			if (h.bin[x * 3 + 1] > cbStrictThreshold)
				cbCount = strictRange;
			else
				cbCount--;

			if (cbCount > 0)
				strictMap.decision[x * 3 + 1] = false;

			if (h.bin[x * 3 + 2] > yStrictThreshold)
				yCount = strictRange;
			else
				yCount--;

			if (yCount > 0)
				strictMap.decision[x * 3 + 2] = false;
		}

		//Perform the first loose decision map generation pass
		crCount = 0;
		cbCount = 0;
		yCount = 0;
		for (int x = 0; x < NUM_BINS; x++)
		{
			if (h.bin[x * 3] > crLooseThreshold)
				crCount = looseRange;
			else
				crCount--;

			if (crCount > 0)
				looseMap.decision[x * 3] = false;
			else
				looseMap.decision[x * 3] = true;

			if (h.bin[x * 3 + 1] > cbLooseThreshold)
				cbCount = looseRange;
			else
				cbCount--;

			if (cbCount > 0)
				looseMap.decision[x * 3 + 1] = false;
			else
				looseMap.decision[x * 3 + 1] = true;

			if (h.bin[x * 3 + 2] > yLooseThreshold)
				yCount = looseRange;
			else
				yCount--;

			if (yCount > 0)
				looseMap.decision[x * 3 + 2] = false;
			else
				looseMap.decision[x * 3 + 2] = true;
		}

		//Perform the second loose decision map generation pass
		crCount = 0;
		cbCount = 0;
		yCount = 0;
		for (int x = NUM_BINS - 1; x >= 0; x--)
		{
			if (h.bin[x * 3] > crLooseThreshold)
				crCount = looseRange;
			else
				crCount--;

			if (crCount > 0)
				looseMap.decision[x * 3] = false;

			if (h.bin[x * 3 + 1] > cbLooseThreshold)
				cbCount = looseRange;
			else
				cbCount--;

			if (cbCount > 0)
				looseMap.decision[x * 3 + 1] = false;

			if (h.bin[x * 3 + 2] > yLooseThreshold)
				yCount = looseRange;
			else
				yCount--;

			if (yCount > 0)
				looseMap.decision[x * 3 + 2] = false;
		}

		//Transfer the strict decision map to global memory
		for (int x = 0; x < NUM_BINS; x++)
		{
			strictMaps[i].decision[x * 3] = strictMap.decision[x * 3];
			strictMaps[i].decision[x * 3 + 1] = strictMap.decision[x * 3 + 1];
			strictMaps[i].decision[x * 3 + 2] = strictMap.decision[x * 3 + 2];
		}

		//Transfer the loose decision map to global memory
		for (int x = 0; x < NUM_BINS; x++)
		{
			looseMaps[i].decision[x * 3] = looseMap.decision[x * 3];
			looseMaps[i].decision[x * 3 + 1] = looseMap.decision[x * 3 + 1];
			looseMaps[i].decision[x * 3 + 2] = looseMap.decision[x * 3 + 2];
		}

		//Transfer the histogram to global memory
		for (int x = 0; x < NUM_BINS; x++)
		{
			histograms[i].bin[x * 3] = h.bin[x * 3];
			histograms[i].bin[x * 3 + 1] = h.bin[x * 3 + 1];
			histograms[i].bin[x * 3 + 2] = h.bin[x * 3 + 2];
		}
	}
}

__global__ void fd_cleanup(unsigned char* mask, int width, int height, int size)
{
	//get the yIndex and xIndex associated with this pixel
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//guard against pixels outside of the image
	if (xIndex < width && yIndex < height)
	{
		//get the image index associated with this pixel
		int i = xIndex + yIndex * width;

		//if this pixel is currently white...
		if (mask[i])
		{
			int whiteCount = 0;
			int blackCount = 0;
			for (int y = -1 * size; y <= size; y++)
			{
				for (int x = -1 * size; x <= size; x++)
				{
					//if this stencil pixel is valid
					if (xIndex + x < width && xIndex + x >= 0 && yIndex + y < height && yIndex + y >= 0)
					{
						//get the stencil pixel's index
						int j = (xIndex + x) + (yIndex + y) * width;

						//if the stencil pixel is white, increment white count
						if (mask[j])
							whiteCount++;
						else //Else increment black count
							blackCount++;
					}
				}
			}

			//syncronize the threads
			syncthreads();

			//move this pixel to background if too small of an object
			if (whiteCount < blackCount)
			{
				mask[i] = 0;
			}
		}
	}
}

__global__ void fd_fill(unsigned char* strictMask, unsigned char* looseMask, int width, int height, int size)
{
	//get the yIndex and xIndex associated with this pixel
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//guard against pixels outside of the image
	if (xIndex < width && yIndex < height)
	{
		//get the image index associated with this pixel
		int i = xIndex + yIndex * width;

		//if this pixel is strictly background but loosely foreground...
		if (looseMask[i] && !strictMask[i])
		{
			int whiteCount = 0;
			int blackCount = 0;
			for (int y = -1 * size; y <= size; y++)
			{
				for (int x = -1 * size; x <= size; x++)
				{
					//if this stencil pixel is valid
					if (xIndex + x < width && xIndex + x >= 0 && yIndex + y < height && yIndex + y >= 0)
					{
						//get the stencil pixel's index
						int j = (xIndex + x) + (yIndex + y) * width;

						//if the stencil pixel is white, increment white count
						if (strictMask[j])
							whiteCount++;
						else //Else increment black count
							blackCount++;
					}
				}
			}

			//syncronize the threads
			syncthreads();

			//If enough strict pixels where white, move this pixel to foreground
			if (whiteCount >= blackCount)
			{
				strictMask[i] = 255;
			}
		}
	}
}


extern "C" void cuda_fd_classify(unsigned char* image, unsigned char* strictMask, unsigned char* looseMask, decisionMap* strictMaps, decisionMap* looseMaps, int width, int height, dim3 gridSize, dim3 blockSize)
{
	fd_classify << < gridSize, blockSize >> > (image, strictMask, looseMask, strictMaps, looseMaps, width, height);
}

extern "C" void cuda_fd_train(energy* energies, histogram* histograms, decisionMap* strictMaps, decisionMap* looseMaps, int width, int height, int xOffset, int yOffset, float alpha, float strictThreshold, int strictRange, float looseThreshold, int looseRange, dim3 gridSize, dim3 blockSize)
{
	fd_train << < gridSize, blockSize >> > (energies, histograms, strictMaps, looseMaps, width, height, xOffset, yOffset, alpha, strictThreshold, strictRange, looseThreshold, looseRange);
}

extern "C" void cuda_fd_update(unsigned char* image, energy* energies, int width, int height, int yOffset, dim3 gridSize, dim3 blockSize)
{
	fd_update << < gridSize, blockSize >> > (image, energies, width, height, yOffset);
}

extern "C" void cuda_fd_cleanup(unsigned char* mask, int width, int height, int size, dim3 gridSize, dim3 blockSize)
{
	fd_cleanup << < gridSize, blockSize >> > (mask, width, height, size);
}

extern "C" void cuda_fd_fill(unsigned char* strictMask, unsigned char* looseMask, int width, int height, int size, dim3 gridSize, dim3 blockSize)
{
	fd_fill << < gridSize, blockSize >> > (strictMask, looseMask, width, height, size);
}
